#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__global__ void add(int *d_a,int *d_b,int *d_c){
	*d_c = *d_a + *d_b;
}

int main(void){
	int a, b, c;
	int *d_c, *d_b, *d_a;
	int size = sizeof(int);

	a = 4; 
	b = 6;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	add<<<1,1>>>(d_a, d_b, d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	cout << c << endl;
	return 0;
}
