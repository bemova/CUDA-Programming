#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
#define N 20
#define BLOCK_DIM 10

void random_inits(int a[N][N]){
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			a[i][j] = rand() % 10;
		}
	}
}

__global__ void add(int a[N][N], int b[N][N], int c[N][N]){
	int i = threadIdx.x;
	int j = threadIdx.y;
	c[i][j] = a[i][j] + b[i][j];
}


void show(int a[N][N], int b[N][N], int c[N][N]){
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			cout << "matrix[" << i << "][" << j << "]" << " = " << a[i][j] << " + " << b[i][j] << "=" << c[i][j] <<"\t";
		}
		cout << endl;
	}
}

int main(void){
	int a[N][N];
	int b[N][N];
	int c[N][N];
	random_inits(a);
	random_inits(b);

	int (*d_a)[N], (*d_b)[N], (*d_c)[N];

	hipMalloc((void**)&d_a, (N*N)*sizeof(int));
	hipMalloc((void**)&d_b, (N*N)*sizeof(int));
	hipMalloc((void**)&d_c, (N*N)*sizeof(int));

	hipMemcpy(d_a, a, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, (N*N)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(N,N);
	add<<<numBlocks,threadsPerBlock>>>(d_a,d_b,d_c);

	hipMemcpy(c, d_c, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	show(a, b, c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	cout<< endl;

	return 0;
}
