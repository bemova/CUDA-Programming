#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define N 10

void random_ints(int *a, int n){
	for (int i = 0; i < n; i++){
		a[i] = rand() % 10;
	}
}

__global__ void add(int *a, int *b, int *c){
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size); 

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add << <1, N >> > (d_a, d_b, d_c);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++){
		cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	free(a); free(b); free(c);
	hipFree(a); hipFree(b); hipFree(c);

}
