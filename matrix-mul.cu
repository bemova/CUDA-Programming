#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

#define N 3
#define M 3
#define K 3
#define BLOCK_SIZE 16

void initialize_matrix(int* matrix, int row, int col){
	for (int i = 0; i < row; ++i) {
		for (int j = 0; j < col; ++j) {
			matrix[i * col + j] = rand() % 10;
			cout << matrix[i * col + j] << "\t";
		}
	}
}

__global__ void mul(int *a, int *b, int *c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			sum += a[row * n + i] * b[i * k + col];
		}
		c[row * k + col] = sum;
	}
}

int main(void) {
	int *a, *b, *c;
	int first_size = sizeof(int) * M * N;
	int second_size = sizeof(int) * N * K;
	int third_size = sizeof(int) * M * K;

	a = (int *)malloc(first_size);
	b = (int *)malloc(second_size);
	c = (int *)malloc(third_size);
	
	initialize_matrix(a, M, N);
	cout << endl;

	initialize_matrix(b, N, K);
	cout << endl;

	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, first_size);
	hipMalloc((void **)&d_b, second_size);
	hipMalloc((void **)&d_c, third_size);

	hipMemcpy(d_a, a, first_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, second_size, hipMemcpyHostToDevice);

	unsigned int grid_rows = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	mul << <dimGrid, dimBlock >> >(d_a, d_b, d_c, M, N, K);
	hipMemcpy(c, d_c, third_size, hipMemcpyDeviceToHost);

	for (int i = 0; i < M * K; i++) {
		cout << c[i] << endl;
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

