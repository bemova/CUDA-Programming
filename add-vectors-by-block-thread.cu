#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define N 50
#define M 10

void random_ints(int *a, int n){
	for (int i = 0; i < n; i++){
		a[i] = rand() % 10;
	}
}

__global__ void add(int *a, int *b, int *c){
	int index = threadIdx.x + blockIdx.x * 10;
	c[index] = a[index] + b[index];
}

int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size); 

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add << <N/M,M >> >(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++){
		cout << i << " -> " << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	free(a), free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
