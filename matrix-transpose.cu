#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

#define M 3
#define N 2
#define BLOCK_SIZE 16

void initialize_matrix(int* matrix, int row, int col){
	for (int i = 0; i < row; ++i) {
		for (int j = 0; j < col; ++j) {
			matrix[i * col + j] = rand() % 10;
			cout << matrix[i * col + j] << "\t";
		}
	}
}

__global__ void transpose(int* input, int* output, int row, int col)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < col && idy < row)
	{
		int pos = idy * col + idx;
		int trans_pos = idx * row + idy;
		output[trans_pos] = input[pos];
	}
}


int main(void) {
	int *a, *b;
	int first_size = sizeof(int) * M * N;
	int second_size = sizeof(int) * N * M;

	a = (int *)malloc(first_size);
	b = (int *)malloc(second_size);

	initialize_matrix(a, M, N);
	cout << endl;


	int *d_a, *d_b;
	hipMalloc((void **)&d_a, first_size);
	hipMalloc((void **)&d_b, second_size);

	hipMemcpy(d_a, a, first_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, second_size, hipMemcpyHostToDevice);

	unsigned int grid_rows = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	transpose << <dimGrid, dimBlock >> >(d_a, d_b, M, N);
	hipMemcpy(b, d_b, second_size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N * M; i++) {
		cout << b[i] << endl;
	}

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}

